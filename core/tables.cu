#include "hip/hip_runtime.h"
#include "tables.cuh"
#include <time.h>
#include <stdio.h>

#include <list>
#include <string> 

#include <hip/hip_runtime.h>
#include <>

#include "assert.hpp"

namespace tables {
	void init() {
		init_table_nbits_values();
		init_table_stdp_values();
		init_table_rand_values();
	}

	const uint8_t const8 = 8;
	rgstr8_t tact_inc8(int& inc) {
		rgstr8_t h_inc8 = 0;

		if (inc == const8 * const8 * const8 * const8 * const8 * const8 * const8 * const8) {
			inc = 0;
		}

		if (inc == 0) {
			inc++;
			return h_inc8;
		}

		if (inc % const8 == 0) {
			h_inc8 = h_inc8 | 0b00000001;

			if (inc % (const8 * const8) == 0) {
				h_inc8 = h_inc8 | 0b00000010;

				if (inc % (const8 * const8 * const8) == 0) {
					h_inc8 = h_inc8 | 0b00000100;

					if (inc % (const8 * const8 * const8 * const8) == 0) {
						h_inc8 = h_inc8 | 0b00001000;

						if (inc % (const8 * const8 * const8 * const8 * const8) == 0) {
							h_inc8 = h_inc8 | 0b00010000;

							if (inc % (const8 * const8 * const8 * const8 * const8 * const8) == 0) {
								h_inc8 = h_inc8 | 0b00100000;

								if (inc % (const8 * const8 * const8 * const8 * const8 * const8 * const8) == 0) {
									h_inc8 = h_inc8 | 0b01000000;

									if (inc % (const8 * const8 * const8 * const8 * const8 * const8 * const8 * const8) == 0) {
										h_inc8 = h_inc8 | 0b10000000;
									}
								}
							}
						}
					}
				}
			}
		}

		inc++;

		return h_inc8;
	}

	const int sz_stdp_values = 128 * 128;
	__constant__ int8_t stdp_values_table[sz_stdp_values];
	__device__ float stdp(const uint8_t cell_spikes,
		const uint8_t synapse_spikes)
	{
		if (cell_spikes < 128)
			if (synapse_spikes < 128) {
				int stdp_index = (int)cell_spikes
					+ ((int)128) * synapse_spikes;
				return ((float)stdp_values_table[stdp_index]) / 14;
			}
		return 0;
	}
	float comp_stdp(const uint8_t cell_spikes,
		const uint8_t synapse_spikes)
	{
		float sum_dt = 0;

		for (int i = 0; i < 8; ++i) {
			if (cell_spikes & (1 << i)) {
				for (int j = 0; j < 8; ++j) {
					if (synapse_spikes & (1 << j)) {
						float d = (float)j - i;
						if (d == 0) d = 0.5;

						d = 1 / d;

						sum_dt += (7 - i) * d;
					}
				}
			}
		}

		return sum_dt / 7;
	};
	void init_table_stdp_values() {
		if (int8_t* buf = (int8_t*)malloc(sz_stdp_values)) {
			for (int i = 0; i < sz_stdp_values; i++) {
				uint8_t x = i % 128;
				uint8_t y = i / 128;

				float val = comp_stdp(x, y);

				buf[x + 128 * y] = (int8_t)val;
			}
			assert_err(hipMemcpyToSymbol(HIP_SYMBOL(stdp_values_table), buf,
				sz_stdp_values * sizeof(int8_t)));

			free(buf);

			console("getted const mem: " + std::to_string(sz_stdp_values));
		}
		else logexit();
	}

	const int sz_nbits_values = 256;
	__constant__ uint8_t nbits_values_table[sz_nbits_values];
	__device__ uint8_t nbits(const uint8_t n) {
		return nbits_values_table[n];
	}
	uint8_t comp_nbits(uint8_t n) {
		uint8_t res = 0;
		while (n) {
			res++;
			n &= n - 1;
		}
		return res;
	}
	void init_table_nbits_values() {
		if (int8_t* buf = (int8_t*)malloc(sz_nbits_values)) {
			for (int i = 0; i < sz_nbits_values; i++)
				buf[i] = comp_nbits(i);

			assert_err(hipMemcpyToSymbol(HIP_SYMBOL(nbits_values_table), buf,
				sz_nbits_values * sizeof(int8_t)));
			free(buf);

			console("getted const mem: " + std::to_string(sz_nbits_values));
		}
		else logexit();
	}

	const uint sz_rand_coseed = 1024;
	__constant__ uint rand_coseed[sz_rand_coseed];
	void init_table_rand_values() {
		if (uint* h_rand_coseed = (uint*)malloc(sz_rand_coseed * sizeof(uint))) {
			srand((uint)time(NULL));

			for (size_t i = 0; i < sz_rand_coseed; i++)
				h_rand_coseed[i] = rand();

			assert_err(hipMemcpyToSymbol(HIP_SYMBOL(rand_coseed), h_rand_coseed,
				sz_rand_coseed * sizeof(uint), 0, hipMemcpyHostToDevice));

			free(h_rand_coseed);

			console("getted const mem: " + std::to_string(sz_rand_coseed));
		}
		else logexit();
	}
	__device__ unsigned int hiprand() {
		unsigned int r = blockIdx.x * blockDim.x + threadIdx.x + clock();
		return rand_coseed[r % sz_rand_coseed];
	}
	__device__ unsigned int static_curand() {
		unsigned int r = blockIdx.x * blockDim.x + threadIdx.x;
		return rand_coseed[r % sz_rand_coseed];
	}
}

namespace dev_const_mem {
	void dev_const_mem::deleter::operator()(void* data) const noexcept {
		std::free(data);
	}

	std::unique_ptr<void, dev_const_mem::deleter> dev_const_mem::make_ptr(std::size_t size) {
		return std::unique_ptr<void, dev_const_mem::deleter>(std::malloc(size));
	}

	const int sz_const_pool = 16384;
	__constant__ uint8_t const_pool_table[sz_const_pool];
	std::list<offset::ptr> parts;

	offset::ptr __add_mempart(void* t, size_t szb) {
		offset::ptr mempart = nullptr;
		if (auto ptr_host_mem = dev_const_mem::make_ptr(szb)) {
			memcpy(ptr_host_mem.get(), t, szb);

			size_t value = 0;
			if (parts.empty()) value = 0;
			else value = parts.back()->value + parts.back()->szb;

			mempart = std::make_shared<offset>(
				dev_const_mem::offset{ std::move(ptr_host_mem), szb, value });

			parts.push_back(mempart);
		}
		else logexit();

		if (uint8_t* temp_table = (uint8_t*)malloc(sz_const_pool)) {
			for (const auto& part : parts) {
				if (!memcpy(&temp_table[part->value], part->hostmem.get(), part->szb))
					logexit();
			}
			assert_err(hipMemcpyToSymbol(HIP_SYMBOL(const_pool_table), temp_table, sz_const_pool));
			free(temp_table);

			void* const_mem_address = nullptr;
			assert_err(hipGetSymbolAddress((void**)&const_mem_address, const_pool_table));

			if (!const_mem_address)
				logexit();

			for (auto& part : parts)
				part->p = (void*)((size_t)const_mem_address + part->value);
		}
		else logexit();

		if (!mempart)
			logexit();

		return mempart;
	}
}

namespace helper {
	__device__  void print(int d) {
		printf("blk %d th %d, %d\n", blockIdx.x, threadIdx.x, d);
	}
	__device__ void print(float f) {
		printf("blk %d th %d, %f\n", blockIdx.x, threadIdx.x, f);
	}
}